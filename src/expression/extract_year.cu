#include "ara/data/column_factories.h"
#include "extract_year.h"

#include <cudf/copying.hpp>
#include <cudf/utilities/error.hpp>

namespace cura::expression::detail {

using cura::data::ColumnVectorCudfColumn;
using cura::data::createCudfColumnVector;
using cura::type::TypeId;

struct DeviceExtractYear {
  __device__ int64_t operator()(int64_t data) {
    return (static_cast<uint64_t>(data) & YEAR_BIT_FIELD_MASK) >>
           YEAR_BIT_FIELD_OFFSET;
  }
};

std::shared_ptr<const Column>
extractYear(const Context &ctx, ThreadId thread_id,
            std::shared_ptr<const ColumnVector> cv,
            const DataType &result_type) {
  CURA_ASSERT(cv->dataType().type_id == TypeId::INT64,
              "Extract year requires int64 operand");
  CURA_ASSERT(result_type.type_id == TypeId::INT64,
              "Extract year's result should be int64");

  auto input = cv->cudf();
  auto output = cudf::allocate_like(
      input, input.size(), cudf::mask_allocation_policy::NEVER,
      ctx.memory_resource->preConcatenate(thread_id));

  if (input.size() == 0) {
    return createCudfColumnVector<ColumnVectorCudfColumn>(result_type,
                                                          std::move(output));
  }

  auto output_view = output->mutable_view();
  if (cv->dataType().nullable) {
    output->set_null_mask(
        rmm::device_buffer{input.null_mask(),
                           cudf::bitmask_allocation_size_bytes(input.size())},
        input.null_count());
  }

  thrust::transform(rmm::exec_policy(0)->on(0), input.begin<int64_t>(),
                    input.end<int64_t>(), output_view.begin<int64_t>(),
                    DeviceExtractYear{});

  CHECK_CUDA(0);

  return createCudfColumnVector<ColumnVectorCudfColumn>(result_type,
                                                        std::move(output));
}

} // namespace cura::expression::detail
